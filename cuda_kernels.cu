#include "hip/hip_runtime.h"
#ifndef OUR_INDEX
#define OUR_INDEX
inline int index(int i, int j) {
    return (i * width) + j;
}
#endif

__global__
void alt_gaussian(int height, int width, float* raw_buffer, float* filter_buffer) {
    float left [5][5] = {
        {2, 4, 5, 4, 2},
        {4, 9, 12, 9, 4},
        {5, 12, 15, 12, 5},
        {4, 9, 12, 9, 4},
        {2, 4, 5, 4, 2}
    };
    float right [5][5];
    float out [5][5];
    int thread_idx = threadIdx.x;
    int stride = blockDim.x;
    for (int i_top = 0; i_top < height; i_top++) {
        for (int j_left = thread_idx; j_left < width; j_left += stride) {

            for (int i = 0; i < 5; i++) {
                for (int j = 0; j < 5; j++) {
                    right[i][j] = raw_buffer[index(i_top + i, j_left + j)];
                }
            }

            for (int i = 2; i < 3; i++) {
                for (int j = 2; j < 3; j++) {
                    out[i][j] = left[i][0] * right[0][j] + left[i][1] * right[1][j] + left[i][2] * right[2][j] + left[i][3] * right[3][j] + left[i][4] * right[4][j];
                }
            }

            filter_buffer[index(i_top + 2, j_left + 2)] = out[2][2];
        }
    }
}
